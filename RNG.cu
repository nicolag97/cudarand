#include "hip/hip_runtime.h"
#include <cmath>

#include "RNG.cuh"

using namespace std;


// Constructor and destructor
__device__ __host__ RNG_Tausworthe::RNG_Tausworthe(){
	_Seed = 129;
}
__device__ __host__ RNG_Tausworthe::RNG_Tausworthe(const unsigned int seed){
	_Seed = seed;
}

// Private functions
__device__ __host__ unsigned int RNG_Tausworthe::TausStep(){
	int S1 = 23, S2 = 5, S3 = 29;
	unsigned int M = 4294967240UL;
	unsigned int z = this->_Seed;
	
	unsigned b = (((z << S1) ^ z) >> S2);

	this->_Seed = (((z & M) << S3) ^ b);
	return _Seed;		
}

// Virtual functions from base class
__device__ __host__ void RNG_Tausworthe::ResetSeed(){
	_Seed = 129;	
}

__device__ __host__ unsigned int RNG_Tausworthe::GetUnsignedInt(){
	return this->TausStep();
}

__device__ __host__ double RNG_Tausworthe::GetUniform(){
	return 2.3283064365387e-10 * this->GetUnsignedInt();	
}

__device__ __host__ double RNG_Tausworthe::GetGauss(){
	double u = this->GetUniform();
	double v = this->GetUniform();

	return sqrt(-2.*log(u)) * cos(2.*M_PI*v);	
}

__device__ __host__ double RNG_Tausworthe::GetBimodal(){
	double gaussian;

	do{
		gaussian = this->GetGauss();
		
		if(gaussian > 0.)
			return 1.;
		else if(gaussian < 0.)
			return -1.;
	}while(gaussian == 0.);
	
	return -1000.;
}

__device__ __host__ void RNG_Tausworthe::SetInternalState(RNG* supportGenerator){
	unsigned int seed;
	
	do
		seed = supportGenerator->GetUnsignedInt();
	while(seed < 129);
	
	this->_Seed = seed;
}



__device__ __host__ RNG_CombinedGenerator::RNG_CombinedGenerator(const unsigned int seed1, const unsigned int seed2, const unsigned int seed3, const unsigned int seed4){
	this->_SeedLCGS = seed1;
	this->_SeedTaus1 = seed2;
	this->_SeedTaus2 = seed3;
	this->_SeedTaus3 = seed4;
}

__device__ __host__ RNG_CombinedGenerator::RNG_CombinedGenerator(){
	this->_SeedLCGS = 0;
	this->_SeedTaus1 = 129;
	this->_SeedTaus2 = 130;
	this->_SeedTaus3 = 131;
}

__device__ __host__ unsigned int RNG_CombinedGenerator::TausStep1(){
	int S1 = 13, S2 = 19, S3 = 12;
	unsigned int M = 4294967294UL;
	unsigned int z = this->_SeedTaus1;
	
	unsigned b = (((z << S1) ^ z) >> S2);
	
	this->_SeedTaus1 = (((z & M) << S3) ^ b);
	return _SeedTaus1;
}

__device__ __host__ unsigned int RNG_CombinedGenerator::TausStep2(){
	int S1 = 2, S2 = 25, S3 = 4;
	unsigned int M = 4294967288UL;
	unsigned int z = this->_SeedTaus2;
	
	unsigned b = (((z << S1) ^ z) >> S2);

	this->_SeedTaus2 = (((z & M) << S3) ^ b);
	return _SeedTaus2;
}
__device__ __host__ unsigned int RNG_CombinedGenerator::TausStep3(){
	int S1 = 3, S2 = 11, S3 = 17;
	unsigned int M = 4294967280UL;
	unsigned int z = this->_SeedTaus3;
	
	unsigned b = (((z << S1) ^ z) >> S2);

	this->_SeedTaus3 = (((z & M) << S3) ^ b);
	return _SeedTaus3;
}

__device__ __host__ unsigned int RNG_CombinedGenerator::LCGStep(){
	unsigned int z = this->_SeedLCGS;
	unsigned int A = 1664525;
	unsigned int C = 1013904223UL;
	this->_SeedLCGS = (A*z + C);
	return _SeedLCGS;
}

__device__ __host__ unsigned int RNG_CombinedGenerator::HybridTausGenerator(){
	return (
		this->TausStep1() ^
		this->TausStep2() ^
		this->TausStep3() ^
		this->LCGStep()
	);
}

__device__ __host__ void RNG_CombinedGenerator::ResetSeed(){
	this->_SeedLCGS = 0;
	this->_SeedTaus1 = 129;
	this->_SeedTaus2 = 130;
	this->_SeedTaus3 = 131;
}

__device__ __host__ unsigned int RNG_CombinedGenerator::GetUnsignedInt(){
	return this->HybridTausGenerator();
}

__device__ __host__ double RNG_CombinedGenerator::GetUniform(){
	return 2.3283064365387e-10 * this->GetUnsignedInt();
}

__device__ __host__ double RNG_CombinedGenerator::GetGauss(){
	double u = this->GetUniform();
	double v = this->GetUniform();

	return sqrt(-2.*log(u)) * cos(2.*M_PI*v);
}

__device__ __host__ double RNG_CombinedGenerator::GetBimodal(){
	double gaussian;

	do{
		gaussian = this->GetGauss();
		
		if(gaussian > 0.)
			return 1.;
		else if(gaussian < 0.)
			return -1.;
	}while(gaussian == 0.);

	return -1000.;
}

__device__ __host__ void RNG_CombinedGenerator::SetInternalState(RNG* supportGenerator){
	this->_SeedLCGS = supportGenerator->GetUnsignedInt();
	
	unsigned int seedTaus1, seedTaus2, seedTaus3;
	
	do
		seedTaus1 = supportGenerator->GetUnsignedInt();
	while(seedTaus1 < 129);
	
	do
		seedTaus2 = supportGenerator->GetUnsignedInt();
	while(seedTaus2 < 129);
	
	do
		seedTaus3 = supportGenerator->GetUnsignedInt();
	while(seedTaus3 < 129);
	
	this->_SeedTaus1 = seedTaus1;
	this->_SeedTaus2 = seedTaus2;
	this->_SeedTaus3 = seedTaus3;
}
